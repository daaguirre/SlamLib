/**
 * @file occupancy_grid_gpu.cu
 * @author daaguirre
 * @brief
 * @date 2022-08-20
 *
 * @copyright Copyright (c) 2022
 *
 */

#include <cuda_tools/error.h>

#include "occupancy_grid_gpu.h"

namespace slam
{

__host__ OccupancyGridGPU::OccupancyGridGPU(
    const int width,
    const int height,
    const float resolution)
    : m_width(width), m_height(height), m_resolution(resolution)
{
}

// __device__ OccupancyGridGPU::OccupancyGridGPU(
//     const int width,
//     const int height,
//     const float resolution)
//     : m_width(width), m_height(height), m_resolution(resolution)
// {
// }

__host__ void OccupancyGridGPU::setup_gpu_environment()
{
    size_t map_bytes = sizeof(float) * m_width * m_height;
    CUDA_RT_CALL(hipMalloc(&m_d_map_ptr, map_bytes));
    CUDA_RT_CALL(hipMalloc(&m_d_transform_ptr, sizeof(Transform)));
}

__host__ void OccupancyGridGPU::delete_gpu_environment()
{
    CUDA_RT_CALL(hipFree(m_d_map_ptr));
    CUDA_RT_CALL(hipFree(m_d_transform_ptr));
}

__device__ CellState OccupancyGridGPU::check_cell_state(
    const IPose<float> &pose,
    const float occ_thr,
    const float free_thr) const
{
    const IPoint *point_ptr = dynamic_cast<const IPoint *>(&pose);
    return check_cell_state(*point_ptr);
}

CellState OccupancyGridGPU::check_cell_state(
    const IPoint &point,
    const float occ_thr,
    const float free_thr) const

{
    if (point.x < width() && point.x >= 0 && point.y < height() && point.y >= 0)
    {
        // column major order
        int idx = point.x + point.y * m_height;
        float occ_prob = m_d_map_ptr[idx];
        if (occ_prob < occ_thr && occ_prob >= 0)
        {
            return CellState::OCCUPIED;
        }
        else if (occ_prob >= free_thr && occ_prob <= 1.0)
        {
            return CellState::FREE;
        }
        return CellState::UNKNOWN;
    }

    return CellState::OUT_OF_BOUNDS;
}
}  // namespace slam