
#include "particle_filter_gpu.h"
#include "slam_lib/utils/math_helpers.h"

namespace slam
{
template <typename FloatT>
ParticleFilterGPU<FloatT>::ParticleFilterGPU(
    const size_t num_particles,
    typename OccupancyGrid<FloatT>::ConstPtr occ_grid)
{
    init_particles();
    size_t particles_bytes = sizeof(Particle) * num_particles;
    CUDA_RT_CALL(hipMalloc(&this->d_particles, particles_bytes));
    CUDA_RT_CALL(hipMemcpy(this->d_particles, this->m_particles.data(), particles_bytes, hipMemcpyHostToDevice));
}

template <typename FloatT>
void ParticleFilterGPU<FloatT>::init_particles()
{
    m_num_particles_inv = 1.0 / static_cast<FloatT>(this->m_num_particles);
    this->m_particles.resize(this->m_num_particles);

    std::vector<IPoint> free_cells = this->m_occ_grid->get_free_cells();
    size_t n = free_cells.size();

    size_t count = 0;
    for(auto& particle : this->m_particles)
    {
        auto it = free_cells.begin();
        std::advance(it, rand() % n);
        *(static_cast<IPoint*>(&particle)) = *it;

        // yaw in [-pi, pi] range
        particle.yaw = wrap_to_pi_range((rand() / static_cast<FloatT>(RAND_MAX)) * 2 * M_PI);
        particle.weight = m_num_particles_inv;
        ++count;
    }
}

template <typename FloatT>
typename ParticleFilterGPU<FloatT>::Particle ParticleFilterGPU<FloatT>::update(
    const RobotReading<FloatT>& previous_odometry,
    const RobotReading<FloatT>& current_odometry)
{
    return Particle();
}

// template <typename FloatT>
// IPose<FloatT> ParticleFilterGPU<FloatT>::sample_motion_model(
//     const IPose<FloatT>& previous_map_pose,
//     const RobotReading<FloatT>& previous_reading,
//     const RobotReading<FloatT>& current_reading)
// {
// }

}  // namespace slam